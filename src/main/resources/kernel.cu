#include "hip/hip_runtime.h"
// Explosion Boost 9000

extern "C" __global__ void nqfaf(int *ld_arr, int *rd_arr, int *col_arr, int *start_jkl_arr, long *result) {
	// gpu intern indice
	int g_id = blockIdx.x * blockDim.x + threadIdx.x;		// global thread id 
	int l_id = threadIdx.x;  								// local thread id within block
	printf("g_id: %d, l_id: %d\n", g_id, l_id);
	// variables		
	unsigned int L = 1 << (N-1);				// queen at the left border of the board (right border is represented by 1) 										
	// start_jkl_arr contains [6 queens free][5 queens for start][5 queens for i][5 queens for j][5 queens for k][5 queens for l] 
	int start = start_jkl_arr[g_id] >> 20;		
	if(start == 69) {				// if we have a pseudo constellation we do nothing 
		return;
	}
	// printf("[%d] N: %d, startjkl: %d\n", g_id, N, start_jkl_arr[g_id]);
	int j = (start_jkl_arr[g_id] >> 10) & 31;	// queen in last row at position j
	int k = (start_jkl_arr[g_id] >> 5) & 31;	// in row k queen at left border, in row l queen at right border
	int l = start_jkl_arr[g_id] & 31;

	// describe the occupancy of the board 
	unsigned int ld = ld_arr[g_id];				// left diagonals, 1 means occupied
	unsigned int rd = rd_arr[g_id];				// right diagonals, 1 means occupied 
	unsigned int col = ~(L-2) ^ col_arr[g_id];		// columns, 1 means occupied 
	unsigned int ld_mem = 0;				// for memorizing board-leaving diagonals 
	unsigned int rd_mem = 0;

	// jkl_queens occupies the diagonals, that go from bottom row to upper right and upper left 
	// and also the left and right column 
	// in row k only L is free and in row l only 1 is free 
	__shared__ unsigned int jkl_queens[N];
	unsigned int rdiag = (L >> j) | (L >> (N-1-k));		// the rd from queen j and k with respect to the last row
	unsigned int ldiag = (L >> j) | (L >> l);		// the ld from queen j and l with respect to the last row
	if(l_id == 0) {
		for(int a = 0;a < N; a++){			// we also occupy the left and right border 
			jkl_queens[N-1-a] = (ldiag >> a) | (rdiag << a) | L | 1;
		}
	}
	ldiag = L >> k;					// ld from queen l with respect to the first row 
	rdiag = 1 << l;					// ld from queen k with respect to the first row 
	if(l_id == 0) {
		for(int a = 0;a < N; a++){
			jkl_queens[a] |= (ldiag << a) | (rdiag >> a);
		}
		jkl_queens[k] = ~L;
		jkl_queens[l] = ~1; 
	}
	__syncthreads();			// avoid corrupt memory behavior 

	ld &= ~(ldiag << start);			// remove queen k from ld 
	if(l != N-1)					// only remove queen k from rd, if no queen in corner (N-1,N-1)
		rd &= ~(rdiag >> start);			// otherwise we continue in row N-1 and find too many solutions 

	// initialize current row as start and solutions as 0
	int row = start;
	unsigned long solutions = 0;

	// calculate the occupancy of the first row
	unsigned int free = ~(ld | rd | col | jkl_queens[row]);	// free is 1 if a queen can be set at the queens location
	unsigned int queen = -free & free;			// the queen that will be set in the current row
	// each row of queens contains the queens of the board of one workitem 
	// local arrays are faster 
	__shared__ unsigned int queens[WORKGROUP_SIZE][N];		// for remembering the queens for all rows for all boards in the work-group 
	queens[l_id][start] = queen;			// we already calculated the first queen in the start row 

	// going forward (setting a queen) or backward (removing a queen)? 										
	int direction = 0;

	// iterative loop representing the recursive setqueen-function 
	// this is the actual solver (via backtracking with Jeff Somers Bit method) 
	// the structure is slightly complicated since we have to take into account the queens at the border, that have already been placed 
	while(row >= start) {				// while we haven't tried everything 
		if(free) {					// if there are free slots in the current row 
			direction = 1;					// we are going forwards 
			queen = -free & free;				// this is the next free slot for a queen (searching from the right border) in the current row
			queens[l_id][row] = queen;			// remember the queen 
			row++;						// increase row counter

			ld_mem = ld_mem << 1 | ld >> 31;		// place the queen in the diagonals and shift them and remember the diagonals leaving the board 
			rd_mem = rd_mem >> 1 | rd << 31;
			ld = (ld | queen) << 1;							
			rd = (rd | queen) >> 1;	
		}
		else {						// if the row is completely occupied 
			direction = 0;					// we are going backwards 
			row--;						// decrease row counter 
			queen = queens[l_id][row];			// recover the queen in order to remove it 
			ld = ((ld >> 1) | (ld_mem << 31)) & ~queen;	// shift diagonals one back, remove the queen and insert the diagonals that had left the board 
			rd = ((rd << 1) | (rd_mem >> 31)) & ~queen;
			ld_mem >>= 1;
			rd_mem <<= 1;						
		}
		free = ~(jkl_queens[row] | ld | rd | col);	// calculate the occupancy of the next row
		free &= ~(queen + direction-1);			// occupy all bits right from the last queen in order to not place the same queen again 
		col ^= queen;					// free up the column AFTER calculating free in order to not place the same queen again		

		if(row == N-1)					// increase the solutions, if we are in the last row 
			solutions++;
	}
	printf("solutions: %d\n", solutions);
	result[g_id] = solutions;			// number of solutions of the work item 
}
